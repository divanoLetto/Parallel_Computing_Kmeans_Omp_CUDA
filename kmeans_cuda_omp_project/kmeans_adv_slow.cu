
#include <hip/hip_runtime.h>
#include <iostream>
#include <numeric>
#include <stdlib.h>
#include <chrono>
#include <random>
#include <stdio.h>
#include <hip/driver_types.h>
#include <hip/hip_runtime_api.h>


#define CUDA_CHECK_RETURN(value) CheckCudaErrorAux(__FILE__,__LINE__, #value, value);

static void CheckCudaErrorAux (const char *, unsigned, const char *, hipError_t);
using namespace std;

static void CheckCudaErrorAux (const char *file, unsigned line, const char *statement, hipError_t err)
{
    if (err == hipSuccess)
        return;
    std::cerr << statement<<" returned " << hipGetErrorString(err) << "("<<err<< ") at "<<file<<":"<<line << std::endl;
    exit (1);
}

struct Record_adv {

    Record_adv(int size, std::vector<float>& x_vector_host, std::vector<float>& y_vector_host){
        size_t size_records = size * sizeof(float);
        CUDA_CHECK_RETURN(hipMalloc(&x, size_records));
        CUDA_CHECK_RETURN(hipMalloc(&y, size_records));
        CUDA_CHECK_RETURN(hipMemcpy(x, x_vector_host.data(), size_records, hipMemcpyHostToDevice));
        CUDA_CHECK_RETURN(hipMemcpy(y, y_vector_host.data(), size_records, hipMemcpyHostToDevice));
    }
    Record_adv(int size){
        std::vector<float>x_vector_host(size, 0);
        std::vector<float>y_vector_host(size, 0);
        size_t size_records = size * sizeof(float);
        CUDA_CHECK_RETURN(hipMalloc(&x, size_records));
        CUDA_CHECK_RETURN(hipMalloc(&y, size_records));
        CUDA_CHECK_RETURN(hipMemcpy(x, x_vector_host.data(), size_records, hipMemcpyHostToDevice));
        CUDA_CHECK_RETURN(hipMemcpy(y, y_vector_host.data(), size_records, hipMemcpyHostToDevice));
    }

    ~Record_adv() {
        hipFree(x);
        hipFree(y);
    }

    float * get_x(){
        return x;
    }
    float * get_y(){
        return x;
    }

    float * x{nullptr};
    float * y{nullptr};
};

__device__ float records_adv_distance_slow(float x_1, float y_1, float x_2, float y_2) {
  return sqrt((x_1 - x_2) * (x_1 - x_2) + (y_1 - y_2) * (y_1 - y_2));
}

Record_adv * get_records_from_file_slow(int NUM_RECORDS){

    std::vector<float> host_x;
    std::vector<float> host_y;
    FILE *fptr;
    if ((fptr = fopen("/home/lorenzo/CLionProjects/kmeans_cuda_omp_project/data_generated","r")) == NULL){
        printf("Error! opening file");
        exit(1);
    }
    for(int i=0; i<NUM_RECORDS; i++){
        int id;
        float number_x;
        float number_y;
        fscanf(fptr,"%d %f %f\n", &id, &number_x, &number_y);
        host_x.push_back(number_x);
        host_y.push_back(number_y);
    }
    fclose(fptr);
    Record_adv* records= new Record_adv(NUM_RECORDS, host_x, host_y);
    return records;
}

Record_adv * get_centroids_from_file_slow( int k){

    std::vector<float> host_x;
    std::vector<float> host_y;
    FILE *fptr;
    if ((fptr = fopen("/home/lorenzo/CLionProjects/kmeans_cuda_omp_project/data_centroids","r")) == NULL){
        printf("Error! opening file");
        exit(1);
    }
    for(int i=0; i<k; i++){
        int id;
        float number_x;
        float number_y;
        fscanf(fptr,"%d %f %f\n", &id, &number_x, &number_y);
        host_x.push_back(number_x);
        host_y.push_back(number_y);
    }
    Record_adv* records= new Record_adv(k, host_x, host_y);
    fclose(fptr);
    return records;

}

__global__ void  cluster_assigment_slow(float* records_x,float *records_y,float*centroids_x,float*centroids_y,float*sum_x,float*sum_y,int num_rec,int k,int * sizes_centroid_d, int* assignment_d){

    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= num_rec) return;
    float x = records_x[index];
    float y = records_y[index];

    float best_distance = INFINITY;
    int best_cluster = -1;

    for (int cluster = 0; cluster < k; ++cluster) {
        const float distance = records_adv_distance_slow(x,y,centroids_x[cluster],centroids_y[cluster]);
        if (distance < best_distance) {
            best_distance = distance;
            best_cluster = cluster;
        }
    }
    assignment_d[index]= best_cluster;
    atomicAdd(&sum_x[best_cluster], x);
    atomicAdd(&sum_y[best_cluster], y);
    atomicAdd(&sizes_centroid_d[best_cluster], 1);
}

__global__ void relocations_centroids_slow(float * centroids_x, float* centroids_y, float* sum_x, float* sum_y, int* sizes_centroid_d){
    const int cluster = threadIdx.x;
    int size = max(sizes_centroid_d[cluster], 1);

    centroids_x[cluster] = sum_x[cluster] / size;
    centroids_y[cluster] = sum_y[cluster] / size;

    sizes_centroid_d[cluster] = 0;
    sum_x[cluster]=0;
    sum_y[cluster]=0;
}

__global__ void print_centroids_slow(float * centroids_x, float* centroids_y){
    const int cluster = threadIdx.x;
    if(cluster==0){
        printf("Print centroid\n");
    }
    __syncthreads();
    printf("Cluster id %d has x: %f y: %f\n", cluster, centroids_x[cluster], centroids_y[cluster]);
}

__global__ void print_records_slow(float * records_x, float* records_y){
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    if(index==0){
        printf("Print records\n");
    }
    __syncthreads();
    printf("Record id %d has x: %f y: %f\n", index, records_x[index], records_y[index]);
}

bool close_centroids_cuda_slow(int j,float* old_centroids_x, float*  old_centroids_y, float*  old_centroids_tmp_x,float*  old_centroids_tmp_y,int NUM_CLUSTERS,float MIN_TOLLERANCE){
    if(j==0){
        return false;
    }
    for(int i=0; i<NUM_CLUSTERS; i++){
        float distance = sqrt(pow(old_centroids_x[i]- old_centroids_tmp_x[i],2)+pow(old_centroids_y[i]- old_centroids_tmp_y[i],2));
        if(distance>MIN_TOLLERANCE){
            return false;
        }
    }
    return true;
}

int kmeans_cuda_slow(float ** host_centroids_x, float ** host_centroids_y, float**host_records_x,float ** host_records_y, int**assignment, int* n_cluster, int *n_record,int NUM_RECORDS, int NUM_CLUSTERS, int MAX_ITERATIONS, float MIN_TOLLERANCE, int TPB)
{
    Record_adv * records = get_records_from_file_slow(NUM_RECORDS);
    Record_adv * centroids = get_centroids_from_file_slow(NUM_CLUSTERS);
    Record_adv * sum = new Record_adv(NUM_CLUSTERS);

    int* sizes_centroid_h = new int[NUM_CLUSTERS];
    for(int i=0; i<NUM_CLUSTERS;i++){
        sizes_centroid_h[i]=0;
    }
    int * sizes_centroid_d;
    int * assignment_d;
    CUDA_CHECK_RETURN(hipMalloc((void **)&sizes_centroid_d, NUM_CLUSTERS*sizeof(int)));
    CUDA_CHECK_RETURN(hipMemcpy(sizes_centroid_d, sizes_centroid_h, NUM_CLUSTERS*sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMalloc((void **)&assignment_d, NUM_RECORDS*sizeof(int)));

    hipDeviceSynchronize();

    float * old_centroids_x = new float[NUM_CLUSTERS];
    float * old_centroids_y = new float[NUM_CLUSTERS];
    float * old_centroids_tmp_x = new float[NUM_CLUSTERS];
    float * old_centroids_tmp_y = new float[NUM_CLUSTERS];
    for(int y=0; y< NUM_CLUSTERS; y++){
        old_centroids_x[y]=0;
        old_centroids_y[y]=0;
    }

    for(int i=0; i<MAX_ITERATIONS; i++){

        cluster_assigment_slow<<<(NUM_RECORDS+TPB-1)/TPB,TPB>>>(records->x,
                records->y,
                centroids->x,
                centroids->y,
                sum->x,
                sum->y,
                NUM_RECORDS,
                NUM_CLUSTERS,
                sizes_centroid_d, assignment_d);
        hipDeviceSynchronize();

        //controllo sullo spostamento dei centroidi: se sotto una soglia termina l'algoritmo
        CUDA_CHECK_RETURN(hipMemcpy(old_centroids_tmp_x, centroids->x, NUM_CLUSTERS * sizeof(float), hipMemcpyDeviceToHost));
        CUDA_CHECK_RETURN(hipMemcpy(old_centroids_tmp_y, centroids->y, NUM_CLUSTERS * sizeof(float), hipMemcpyDeviceToHost));
        if(close_centroids_cuda_slow(i,old_centroids_x, old_centroids_y, old_centroids_tmp_x, old_centroids_tmp_y,NUM_CLUSTERS, MIN_TOLLERANCE)) {
            break;
        }
        for (int j=0; j<NUM_CLUSTERS; j++){
            old_centroids_x[j]=old_centroids_tmp_x[j];
            old_centroids_y[j]=old_centroids_tmp_y[j];
        }


        relocations_centroids_slow<<<1, NUM_CLUSTERS>>>(centroids->x,
                centroids->y,
                sum->x,
                sum->y,
                sizes_centroid_d );

        hipDeviceSynchronize();
    }

    *n_cluster = NUM_CLUSTERS;
    *n_record = NUM_RECORDS;
    *assignment = new int[NUM_RECORDS];
    *host_centroids_x = new float[NUM_CLUSTERS];
    *host_centroids_y = new float[NUM_CLUSTERS];

    CUDA_CHECK_RETURN(hipMemcpy(*assignment, assignment_d, NUM_RECORDS* sizeof(int), hipMemcpyDeviceToHost));

    size_t size_clusters = NUM_CLUSTERS * sizeof(float);
    CUDA_CHECK_RETURN(hipMemcpy(*host_centroids_x, centroids->x, size_clusters, hipMemcpyDeviceToHost));
    CUDA_CHECK_RETURN(hipMemcpy(*host_centroids_y, centroids->y, size_clusters, hipMemcpyDeviceToHost));

    *host_records_x= new float[NUM_RECORDS];
    *host_records_y= new float[NUM_RECORDS];

    size_t size_records = NUM_RECORDS * sizeof(float);
    CUDA_CHECK_RETURN(hipMemcpy(*host_records_x, records->x, size_records, hipMemcpyDeviceToHost));
    CUDA_CHECK_RETURN(hipMemcpy(*host_records_y, records->y, size_records, hipMemcpyDeviceToHost));
    records->~Record_adv();
	return 0;
}



